#include "hip/hip_runtime.h"
#include "mex.h"
#include "matrix.h"
#include <stdio.h>
#include <math.h>
#include <limits.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__
void calcresKernel(double *resi, double *P_mat_r, double *P_mat_i, double *i_mage_r, double *i_mage_i, int xsize, int ysize, int nr_fms, int nr_kr, int nr_echo)
{
    //extern __shared__ double s_data[];

    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int ifms = threadIdx.z + blockDim.z * blockIdx.z;

    int ikr,iecho,k,l;
    double sum, imagsum, realsum, a1, a2, b1, b2;
    double minimum;    
    
    /*
    for (int ism1 = 0; ism1 < nr_echo; ism1++)
    {
        s_data[ism1*2] = i_mage_r[ix+iy*xsize+ism1*xsize*ysize];
        s_data[ism1*2 + 1] = i_mage_i[ix+iy*xsize+ism1*xsize*ysize];
    }
    __syncthreads;
    */
    
    //Initialize with maximum double 
    minimum = DBL_MAX;
    
    if (ix < xsize && iy < ysize && ifms < nr_fms)  
    {
        for (ikr=0; ikr<nr_kr; ikr++){
            sum = 0;

            // sum is calculated over k and l
            for (k=0;k<nr_echo;k++){
                realsum = 0;
                imagsum = 0;

                for (l=0;l<nr_echo;l++){
    //                          sum = sum +P_mat[k][l][ifms][ikr]*image[ix][iy][l]; but complex multiplication
                    a1 = P_mat_r[k+l*nr_echo+ifms*nr_echo*nr_echo+ikr*nr_echo*nr_echo*nr_fms];
                    b1 = P_mat_i[k+l*nr_echo+ifms*nr_echo*nr_echo+ikr*nr_echo*nr_echo*nr_fms];

                    //a2 = s_data[2*l];
                    //b2 = s_data[2*l + 1];
                    a2 = i_mage_r[ix+iy*xsize+l*xsize*ysize];
                    b2 = i_mage_i[ix+iy*xsize+l*xsize*ysize];
                    realsum = realsum + a1*a2 - b1*b2;
                    imagsum = imagsum + a1*b2 + a2*b1;              
                }
                // calculation of square of absolute value of complex product:
                sum = sum + realsum*realsum + imagsum*imagsum;
            }
            if (sum<minimum){
                minimum = sum;
    //                         printf("sum is smaller!\n");  
            }
        }
        //printf("assigning value to residual\n");
        resi[ifms + ix*nr_fms + iy*nr_fms*xsize] = minimum;
    }
}

static void HandleError( hipError_t err,
                const char *file,
                int line )
{
if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", 
            hipGetErrorString( err ),
            file, line );
    // exit( EXIT_FAILURE );
}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

inline dim3 computeGrid1D(const dim3 &block, const int w)
{
    int num = (w + block.x -1) / (block.x);
    //int num = w + block.x - 1;
    return dim3(num, 1, 1);   // TODO (3.2) compute 1D grid size from block size
}

inline dim3 computeGrid2D(const dim3 &block, const int w, const int h)
{
    int num1 = (w + block.x -1) / (block.x);
    int num2 = (h + block.y -1) / (block.y);
    //int num = 1;
    return dim3(num1, num2, 1);   // TODO (3.2) compute 2D grid size from block size
}

inline dim3 computeGrid3D(const dim3 &block, const int w, const int h, const int s)
{
    int num1 = (w + block.x -1) / (block.x);
    int num2 = (h + block.y -1) / (block.y);
    int num3 = (s + block.z -1) / (block.z);
    //int num = 1;
    return dim3(num1, num2, num3);   // TODO (3.2) compute 3D grid size from block size
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
//example out- and input matrix sizes: [921,400,400] = calcResidualInC(Phi[20,20,921,26],Image[400,400,20])
{
    //====================  Variable definition  ======================
    
    int number_dims_out;
    const mwSize *dim_phi, *dim_array_image;
    mwSize dims_out[] = {1,2,3};
    
    //Inputs
    int num_fms;
    int num_kr;
    int num_echo;
    int xres, yres;
    double *P_matrix_r;
    double *P_matrix_i;
    double *in_image_r;
    double *in_image_i;
    
    //Output
    double *residual; //mxArray could also be a data type
    
    //CUDA arrays
    double *d_P_matrix_r = NULL;
    double *d_P_matrix_i = NULL;
    double *d_in_image_r = NULL;
    double *d_in_image_i = NULL;
    double *d_residual = NULL;
    
    int fmscnt;
    int ix,iy,ifms,nx;
    
    //==============================================================
    
    //for output initialization, 3 dimensions are created
    number_dims_out = 3;
    //size of Psi matrix is extracted: 20x20x921x26
    dim_phi = mxGetDimensions(prhs[0]);
    num_fms = dim_phi[2];
    num_kr = dim_phi[3];
    //size of input image is extracted: 400x400x20 xres*yres*nrecho
    dim_array_image = mxGetDimensions(prhs[2]);
    num_echo = dim_array_image[2];
    xres = dim_array_image[0];
    yres = dim_array_image[1];  
    
    // ============= Data Initialization and Allocation ==============
    //Output
    dims_out[0] = num_fms;
    dims_out[1] = xres;
    dims_out[2] = yres;
    plhs[0] = mxCreateNumericArray(number_dims_out, dims_out, mxDOUBLE_CLASS, mxREAL);
    residual = mxGetPr(plhs[0]);
    
    //Get input data
    P_matrix_r = mxGetPr(prhs[0]);
    P_matrix_i = mxGetPr(prhs[1]);
    in_image_r = mxGetPr(prhs[2]);
    in_image_i = mxGetPr(prhs[3]);
    
    // Allocate cuda memory for CUDA arrays
    HANDLE_ERROR(hipMalloc( &d_P_matrix_r, dim_phi[0]*dim_phi[1]*dim_phi[2]*dim_phi[3]*sizeof(double))); //CUDA_CHECK;
    HANDLE_ERROR(hipMalloc( &d_P_matrix_i, dim_phi[0]*dim_phi[1]*dim_phi[2]*dim_phi[3]*sizeof(double))); //CUDA_CHECK;
    HANDLE_ERROR(hipMalloc( &d_in_image_r, num_echo*xres*yres*sizeof(double))); //CUDA_CHECK;
    HANDLE_ERROR(hipMalloc( &d_in_image_i, num_echo*xres*yres*sizeof(double))); //CUDA_CHECK;
    HANDLE_ERROR(hipMalloc( &d_residual, num_fms*xres*yres*sizeof(double))); //CUDA_CHECK;
    
    //Copy memory
    HANDLE_ERROR(hipMemcpy(d_P_matrix_r, P_matrix_r, dim_phi[0]*dim_phi[1]*dim_phi[2]*dim_phi[3]*sizeof(double), hipMemcpyHostToDevice)); //CUDA_CHECK; 
    HANDLE_ERROR(hipMemcpy(d_P_matrix_i, P_matrix_i, dim_phi[0]*dim_phi[1]*dim_phi[2]*dim_phi[3]*sizeof(double), hipMemcpyHostToDevice)); //CUDA_CHECK;
    HANDLE_ERROR(hipMemcpy(d_in_image_r, in_image_r, num_echo*xres*yres*sizeof(double), hipMemcpyHostToDevice)); //CUDA_CHECK;
    HANDLE_ERROR(hipMemcpy(d_in_image_i, in_image_i, num_echo*xres*yres*sizeof(double), hipMemcpyHostToDevice)); //CUDA_CHECK;
//     hipMemcpy(d_residual, residual, num_fms*xres*yres*sizeof(double), hipMemcpyHostToDevice)); //CUDA_CHECK;
    
    //================ Calculation of residual function ============
    //calcres(residual,P_matrix_r,P_matrix_i,in_image_r,in_image_i,xres,yres,num_fms,num_kr,num_echo);
    
    // calculate block and grid size
    dim3 block = dim3(8, 8, 16);     // Specify suitable block size
    dim3 grid = computeGrid3D(block, xres, yres, num_fms);
    size_t smBytes = 2 * num_echo * sizeof(double) ;
    //printf("%d, %d, %d", grid.x, grid.y, grid.z);

    // run CUDA kernel
    calcresKernel <<<grid,block>>> (d_residual, d_P_matrix_r, d_P_matrix_i, d_in_image_r, d_in_image_i,xres,yres,num_fms,num_kr,num_echo); //CUDA_CHECK;
    HANDLE_ERROR(hipMemcpy(residual, d_residual, num_fms*xres*yres*sizeof(double), hipMemcpyDeviceToHost)); //CUDA_CHECK;
    
    #if MX_HAS_INTERLEAVED_COMPLEX
        //printf("it has interleave!");
    #endif 
    
    HANDLE_ERROR(hipFree(d_P_matrix_r));
    HANDLE_ERROR(hipFree(d_P_matrix_i));
    HANDLE_ERROR(hipFree(d_in_image_r));
    HANDLE_ERROR(hipFree(d_in_image_i));
    HANDLE_ERROR(hipFree(d_residual));
    
//     delete[] imgIn;
//     delete[] imgOut;    
        
//     do not call mxDestroyArray or mxFree on an mxArray returned in a left-side argument of a mex-file
//     mxDestroyDouble(in_image);
//     mxDestroyDouble(P_matrix);
        
}